#include "hip/hip_runtime.h"

//--------------------------------------------------------------------------------
// NVIDIA(R) GVDB VOXELS
// Copyright 2017, NVIDIA Corporation
//
// Redistribution and use in source and binary forms, with or without modification, 
// are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright notice, this 
//    list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright notice, this 
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
// 3. Neither the name of the copyright holder nor the names of its contributors may 
//    be used to endorse or promote products derived from this software without specific 
//   prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, 
// SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT 
// OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) 
// HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
// TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, 
// EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
// 
// Version 1.0: Rama Hoetzlein, 5/1/2017
//----------------------------------------------------------------------------------

#include "optix_extra_math.cuh"

#define REFLECT_DEPTH	1
#define REFRACT_DEPTH	2
#define SHADOW_DEPTH	5

#define ANY_RAY			0
#define	SHADOW_RAY		1
#define VOLUME_RAY		2
#define MESH_RAY		3
#define REFRACT_RAY		4

struct Material {
	char		name[64];
	int			id;
	float		light_width;		// light scatter
	
	float3		amb_color;
	float3		env_color;			// 0.5,0.5,0.5
	float3		diff_color;			// .6,.7,.7
	float3		spec_color;			// 3,3,3
	float		spec_power;			// 400		

	float		shadow_width;		// shadow scatter
	float		shadow_bias;

	float		refl_width;			// reflect scatter
	float3		refl_color;			// 1,1,1		
	float		refl_bias;

	float		refr_width;			// refract scatter
	float		refr_ior;			// 1.2
	float3		refr_color;			// .35, .4, .4
	float		refr_amount;		// 10
	float		refr_offset;		// 15
	float		refr_bias;
};

rtDeclareVariable(float3,       light_pos, , );
rtDeclareVariable(Material,		mat, , );

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(unsigned int, shadow_enable, , );
rtDeclareVariable(unsigned int, mirror_enable, , );
rtDeclareVariable(unsigned int, cone_enable, , );
rtDeclareVariable(int,          max_depth, , );

rtDeclareVariable(float3,		shading_normal,		attribute shading_normal, ); 
rtDeclareVariable(float3,		front_hit_point,	attribute front_hit_point, );
rtDeclareVariable(float3,		back_hit_point,		attribute back_hit_point, );
rtDeclareVariable(float4,		deep_color,			attribute deep_color, );
rtDeclareVariable(int,			obj_type,			attribute obj_type, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2,        launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, sample, , );

rtBuffer<unsigned int, 2>       rnd_seeds;

rtTextureSampler<float4, 2>		envmap;

struct RayInfo
{
	float3	result;
	float	length; 
	float	alpha;
	int		depth;
	int		rtype;
	float   importance;
};

rtDeclareVariable(RayInfo, rayinfo, rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay (float3 origin, float3 direction, int depth, int rtype, float& length )
{
  optix::Ray ray = optix::make_Ray( origin, direction, 0, 0.0f, RT_DEFAULT_MAX );
  RayInfo rayi;
  rayi.length = 0.f;
  rayi.depth = depth;
  rayi.rtype = rtype;
  rayi.alpha = 1.f;
  rayi.importance = 1.f;
  rtTrace( top_object, ray, rayi );
  length = rayi.length;
  return (rtype == SHADOW_RAY) ? make_float3(rayi.alpha, rayi.alpha, rayi.alpha) : rayi.result;
}

static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

// -----------------------------------------------------------------------------

float3 __device__ __inline__ jitter_sample ()
{	 
	uint2 index = make_uint2(launch_index.x & 0x7F, launch_index.y & 0x7F);
	unsigned int seed = rnd_seeds[index];  	
	float uu = rnd(seed) - 0.5f;
	float vv = rnd(seed) - 0.5f;
	float ww = rnd(seed) - 0.5f;
	rnd_seeds[index] = seed;
	return make_float3(uu, vv, ww);
}

float3 __device__ __inline__ sampleEnv(float3 dir)
{
	float u = atan2f(dir.x, dir.z) * M_1_PIf;
	float v = 1.0 - dir.y;
	return (v < 0) ? make_float3(.1, .1, .1) : make_float3( tex2D(envmap, u, v) );
}

RT_PROGRAM void trace_surface ()
{
	// parameters

	float importance_cutoff = 1e-2f;
	float3 cutoff_color = make_float3(0.34f, 0.55f, 0.85f);
	float fresnel_exponent = 3.0f;
	float fresnel_minimum = 0.1f;
	float fresnel_maximum = 1.0f;
	float refraction_index = 1.4f;
	float3 refraction_color = make_float3(1.0f, 1.0f, 1.0f);
	float3 reflection_color = make_float3(1.0f, 1.0f, 1.0f);
	int refraction_maxdepth = 100;
	int reflection_maxdepth = 100;
	float3 extinction_constant = make_float3(__logf(0.80f), __logf(0.89f), __logf(0.75f));
	float3 shadow_attenuation = make_float3(0.4f, 0.7f, 0.4f);
	
	// geometry vectors
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal  
	const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
	const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
	const float3 raydir = ray.direction;                                            // incident direction
	float3 lightdir, spos, refldir, refrdir, reflclr, refrclr, shadowclr;
	float3 jit = jitter_sample();
	float ndotl, refldist, refrdist;

	if (isnan(fhp.x) || isnan(raydir.x) ) return;

	float d = length(fhp - ray.origin);	
	lightdir = normalize(normalize(light_pos - fhp) + jit * mat.light_width );
	ndotl = dot(n, lightdir);

	// shading			
	float3 diffuse		= mat.diff_color * sampleEnv ( lightdir ) * max(0.0f, ndotl );
	float3 spec			= mat.spec_color * pow( max(0.0f, dot( n, normalize(-raydir + normalize(light_pos-fhp)))), (float) mat.spec_power );
	
	reflclr = make_float3(0, 0, 0);
	refrclr = make_float3(0, 0, 0);
	shadowclr = make_float3(1, 1, 1);

	if (rayinfo.depth < REFLECT_DEPTH && mat.refl_width > 0) {			
		// reflection sample					
		refldir = normalize(normalize(2 * dot(n, -raydir) * n + raydir) + jit * mat.refl_width);
		reflclr = TraceRay(fhp + refldir*mat.refl_bias, refldir, rayinfo.depth + 1, ANY_RAY, refldist) * mat.refl_color;
	}
	
	if (rayinfo.depth < REFRACT_DEPTH && mat.refr_width > 0) {
		// refraction sample
		optix::refract(refrdir, raydir, n, mat.refr_ior);
		refrdir = normalize(refrdir);// + jit * mat.refr_width);
		if (!isnan(refrdir.x)) {
			refrclr = TraceRay(fhp + refrdir*mat.refr_bias, refrdir, rayinfo.depth + 1, REFRACT_RAY, refrdist);
			// refrclr = lerp3(mat.refr_color, refrclr, refrdist / mat.refr_offset);			
			// refrclr = lerp3(refrclr*mat.refr_amount, mat.refr_color,  min(1.0f, refrdist / mat.refr_offset) );
		}
	}
	// if (rayinfo.depth < SHADOW_DEPTH) {
	// 	// shadow sample		
	// 	for (int i = 0; i < 2; i++) {
	// 		lightdir = normalize(normalize(light_pos - fhp) + jitter_sample() * mat.light_width);
	// 		shadowclr *= TraceRay(fhp + lightdir*mat.shadow_bias, lightdir, rayinfo.depth + 1, SHADOW_RAY, refldist);
	// 	}
	// }
	if (mat.env_color.x == 1) {
		float chk = ((int(floor(fhp.x / mat.env_color.y) + floor(fhp.z / mat.env_color.y)) & 1) == 0) ? 1.0 : mat.env_color.z;
		diffuse *= chk;

	}

	rayinfo.result = (diffuse*make_float3(.85f, .85f, .85f) + spec + mat.amb_color)*shadowclr.x + (reflclr + refrclr)*(shadowclr.x*0.3+0.7);
	// rayinfo.result = reflclr + refrclr;
	// printf("%f,%f,%f\n",raydir.x, raydir.y, raydir.z);
	rayinfo.length = d;
	rayinfo.alpha = .9f;
	// rayinfo.alpha = 0; // deep_color.w;
	// rayinfo.matid = hit_mat;
}


// -----------------------------------------------------------------------------

//
// Attenuates shadow rays for shadowing transparent objects
//

RT_PROGRAM void trace_shadow ()
{
	float3 shadow_attenuation = make_float3(0.4f, 0.4f, 0.4f);
	float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
	float nDi = fabs(dot(world_normal, ray.direction));

	rayinfo.alpha *= (1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1))).x;

	rtIgnoreIntersection();	
}


static __device__ __inline__ float3 schlick( float nDi, const float3& rgb )
{
	float r = fresnel_schlick(nDi, 5, rgb.x, 1);
	float g = fresnel_schlick(nDi, 5, rgb.y, 1);
	float b = fresnel_schlick(nDi, 5, rgb.z, 1);
	return make_float3(r, g, b);
}


//
// (NEW)
// Attenuates shadow rays for shadowing transparent objects
//

RT_PROGRAM void glass_any_hit_shadow()
{
  float3 shadow_attenuation = make_float3(0.4f, 0.7f, 0.4f);
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  rayinfo.alpha *= (1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1))).x;

  rtIgnoreIntersection();			
}


// //
// // Dielectric surface shader
// //
// rtDeclareVariable(float3,       cutoff_color, , );
// rtDeclareVariable(float,        fresnel_exponent, , );
// rtDeclareVariable(float,        fresnel_minimum, , );
// rtDeclareVariable(float,        fresnel_maximum, , );
// rtDeclareVariable(float,        refraction_index, , );
// rtDeclareVariable(int,          refraction_maxdepth, , );
// rtDeclareVariable(int,          reflection_maxdepth, , );
// rtDeclareVariable(float3,       refraction_color, , );
// rtDeclareVariable(float3,       reflection_color, , );
// rtDeclareVariable(float3,       extinction_constant, , );
RT_PROGRAM void glass_closest_hit_radiance()
{
  // intersection vectors
  float importance_cutoff = 1e-2f;
  float3 cutoff_color = make_float3(0.34f, 0.55f, 0.85f);
  float fresnel_exponent = 3.0f;
  float fresnel_minimum = 0.1f;
  float fresnel_maximum = 1.0f;
  float refraction_index = 1.4f;
  float3 refraction_color = make_float3(1.0f, 1.0f, 1.0f);
  float3 reflection_color = make_float3(1.0f, 1.0f, 1.0f);
  int refraction_maxdepth = 100;
  int reflection_maxdepth = 100;
  float3 extinction_constant = make_float3(__logf(0.80f), __logf(0.89f), __logf(0.75f));
  float3 shadow_attenuation = make_float3(0.4f, 0.7f, 0.4f);

	// const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
  const float3 h = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);           // hitpoint
  const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 i = ray.direction;                                            // incident direction

  float reflection = 1.0f;
  float3 result = make_float3(0.0f);

  float3 beer_attenuation;
  if(dot(n, ray.direction) > 0){
    // Beer's law attenuation
    beer_attenuation = exp(extinction_constant * t_hit);
  } else {
    beer_attenuation = make_float3(1);
  }

  // refraction
  if (rayinfo.depth < min(refraction_maxdepth, max_depth))
  {
    float3 t;                                                            // transmission direction
    if ( refract(t, i, n, refraction_index) )
    {

      // check for external or internal reflection
      float cos_theta = dot(i, n);
      if (cos_theta < 0.0f)
        cos_theta = -cos_theta;
      else
        cos_theta = dot(t, n);

      reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

      float importance = rayinfo.importance * (1.0f-reflection) * optix::luminance( refraction_color * beer_attenuation );
      if ( importance > importance_cutoff ) {
        optix::Ray ray( h, t, 0, 0.0f, RT_DEFAULT_MAX  );
        RayInfo refr_prd;
        refr_prd.depth = rayinfo.depth+1;
		refr_prd.importance = importance;
		refr_prd.alpha = 1.0f;
		refr_prd.length = 0;
		refr_prd.rtype = 0;


        rtTrace( top_object, ray, refr_prd );
        result += (1.0f - reflection) * refraction_color * refr_prd.result;
      } else {
        result += (1.0f - reflection) * refraction_color * cutoff_color;
      }
    }
    // else TIR
  }

  // reflection
  if (rayinfo.depth < min(reflection_maxdepth, max_depth))
  {
    float3 r = reflect(i, n);

    float importance = rayinfo.importance * reflection * optix::luminance( reflection_color * beer_attenuation );
    if ( importance > importance_cutoff ) {
      optix::Ray ray( h, r, 0, 0.0f, RT_DEFAULT_MAX  );
      RayInfo refl_prd;
      refl_prd.depth = rayinfo.depth+1;
	  refl_prd.importance = importance;
	  refl_prd.alpha = 1.0f;
	  refl_prd.length = 0;
	  refl_prd.rtype = 0;

      rtTrace( top_object, ray, refl_prd );
      result += reflection * reflection_color * refl_prd.result;
    } else {
      result += reflection * reflection_color * cutoff_color;
    }
  }

  result = result * beer_attenuation;

  rayinfo.result = result;
}